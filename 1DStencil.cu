#include "hip/hip_runtime.h"
__gloabl__.void stencil_1d(int *in, int *out)
{
    __shared__ int temp[BLOCL_SIZE + 2 * RADIUS];
    int gindex = threadIdx.x + blockIdx.x * blockDim.x;
    int lindex = threadIdx.x + RADIUS;

    // Read input elements into shared memory
    temp[lindex] = in[gindex];

    // To fill up the most front and back empty RADIUS memory
    if(threadIdx.x < RADIUS)
    {
        temp[lindex - RADIUS] = in[gindex - RADIUS];
        temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE];
    }

    // To prevent data race
    // Synchronize first
    __syncthreads();

    // Apply the stencil
    int result = 0;
    for(int offset = -RADIUS; offset <=RADIUS; offset++)
    {
        // Did not access global memory, using shared memory
        result += temp[lindex + offset];
    }

    // Store the result
    out[gindex] = result;
}

int main()
{
    return 0;
}