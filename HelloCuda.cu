// __global__ runs on the device, is called from host code
// Device function processed by NVIDIA compiler
// void return type, can pass in any params 

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void mykernel(void)
{

}

// Host functions processed by standard compiler
int main(void)
{
    // <<<>>> marks a call from host code to device code
    // kernal launch
    // return to the parameters (1,1) in a moment
    // 1 block, 1 thread
    mykernel<<<1,1>>>();
    printf("Hello CUDA! \n");
    return 0;
}

// nvcc separates source code into host and device components