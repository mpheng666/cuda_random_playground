
#include <hip/hip_runtime.h>
// Function that runs on the GPU is called a kernel
// add() runs on the device, a,b,c must point to device memory
// May be passed to/from host code, may not be dereference in host code
__global__ void add(int *a, int *b, int *c)
{
    *c = *a + *b;
}

// Memory handling API
// cudaMalloc()
// cudaFree()
// cudaMemcpy()

int main()
{
    int a, b, c;                // Host copies
    int *d_a, *d_b, *d_c;       // Device copies
    int size = sizeof(int);

    // Allocate space for device copies
    // ** because it is a returned value of a pointer (address of pointer)
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    a = 3;
    b = 100;

    // copy inputs to device
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    // add<<<N_times, 1>>>()
    add<<<1,1>>>(d_a, d_b, d_c);

    // copy result back to host
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

    // cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}