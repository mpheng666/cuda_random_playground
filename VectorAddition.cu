
#include <hip/hip_runtime.h>
__global__ void add(int *a, int *b, int *c)
{
    // blockIdc.x magic variabele to access block index
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void random_ints(int* a, int N)
{
   int i;
   for (i = 0; i < N; ++i)
    a[i] = rand();
}

int main()
{
    int N = 512;
    int *a, *b, *c;                // Host copies
    int *d_a, *d_b, *d_c;       // Device copies
    int size = N * sizeof(int);

    // Allocate space for device copies
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    // Allocate space for host copies and setup input values
    a = (int *)malloc(size); random_ints(a, N);
    b = (int *)malloc(size); random_ints(b, N);
    c = (int *)malloc(size);

    // copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // add<<<N_times, 1>>>()
    add<<<N,1>>>(d_a, d_b, d_c);

    // copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // cleanup
    free(a); free(b); free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}