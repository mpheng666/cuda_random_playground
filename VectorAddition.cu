#include "hip/hip_runtime.h"
# define N (2048*2048)
#define THREADS_PER_BLOCK 512

__global__ void add(int *a, int *b, int *c, int n)
{
    // // blockIdc.x magic variabele to access block index
    // c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];

    // // Use thread instead
    // c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];

    // Use parallel threads and parallel blocks
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    // Typical problems are not whole multiples of blockDim.x
    // Avid accessing beyond the end of the arrays
    if(index<n)
    {
        c[index] = a[index] + b[index];
    }
}

void random_ints(int* a, int N)
{
   int i;
   for (i = 0; i < N; ++i)
    a[i] = rand();
}

int main()
{
    int *a, *b, *c;                // Host copies
    int *d_a, *d_b, *d_c;       // Device copies
    int size = N * sizeof(int);

    // Allocate space for device copies
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    // Allocate space for host copies and setup input values
    a = (int *)malloc(size); random_ints(a, N);
    b = (int *)malloc(size); random_ints(b, N);
    c = (int *)malloc(size);

    // copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // add<<<blocks, threads>>>()
    // add<<<N,1>>>(d_a, d_b, d_c);
    add<<<N/THREADS_PER_BLOCK ,THREADS_PER_BLOCK>>>(d_a, d_b, d_c, N);
    // add<<<(N + M-1)/M, M>>>(d_a, d_b, d_c, N);

    // copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // cleanup
    free(a); free(b); free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}